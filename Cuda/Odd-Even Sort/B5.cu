
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<stdlib.h>

__global__ void even(int *a,int n)

{
int k=blockIdx.x*512+threadIdx.x;
k=2*k;
if(k < n-1)
{
if(a[k] > a[k+1])
{
int t=a[k];
a[k]=a[k+1];
a[k+1]=t;

}
}
}

__global__ void odd(int *a,int n)
{
int k=blockIdx.x*512+threadIdx.x;
k=2*k+1;
if(k < n-1)
{
if(a[k] > a[k+1])
{
int t=a[k];
a[k]=a[k+1];
a[k+1]=t;

}
}
}
void odd_even_sort(int *a, const int n)
{

int *ad;
hipMalloc((void **)&ad,n*sizeof(int));
hipMemcpy(ad,a,n*sizeof(int),hipMemcpyHostToDevice);
for(int i=0;i < n/2;i++)
{
even<<<n/1024+1,512>>>(ad,n);
odd<<<n/1024+1,512>>>(ad,n);
}
hipMemcpy(a,ad,n*sizeof(int),hipMemcpyDeviceToHost);
return;
}

int main()
{
int n = 20;
int a[n];
time_t t;
srand((unsigned)time(&t));


int x,flag;
for (unsigned i = 0 ; i < n ; i++)
{
x=rand()%n;
flag=0;
for(int j=0;j < i;j++)
{
if(a[j]==x)
{
i--;
flag=1;
break;
}

}
if(flag==0)
a[i]=x;
}
printf("\n\n original array\n");
for(int i=0;i < n;i++)
printf("%d\t ",a[i]);
printf("\n\n");
odd_even_sort(a,n);
printf("\n\n after sorting\n");
for(int i=0;i < n;i++)
printf("%d\t ",a[i]);


}
