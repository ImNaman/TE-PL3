//Assignment No-B3

 
#include <hip/hip_runtime.h>
#include "iostream"  
 using namespace std;  
 __global__ void sort(int *arr_d, int pivot, int len, int *arrl_d, int *arrh_d)  
 {  
   int id = threadIdx.x;  
   bool flag;  
   int element = arr_d[id+1];  
   if( element <= pivot )  
     flag = true;  
   else  
     flag = false;  
   __syncthreads();  
   if(flag)  
     arrl_d[id] = element;  
   else  
     arrh_d[id] = element;  
 }  
 void quicksort(int *arr, int len)  
 {  
   if(len == 1 || len == 0)  
     return;  
   int pivot = arr[0];  
   size_t size = len*sizeof(int);  
   int *arr_d, *arrl_d, *arrh_d, *arrl, *arrh;  
   arrl = new int[len];  
   arrh = new int[len];  
   for(int i=0; i<len; i++)  
   {  
     arrl[i] = -9999;  
     arrh[i] = -9999;  
   }  
   hipMalloc((void **)&arr_d, size);  
   hipMalloc((void **)&arrl_d, size);  
   hipMalloc((void **)&arrh_d, size);  
   hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice);  
   hipMemcpy(arrl_d, arrl, size, hipMemcpyHostToDevice);  
   hipMemcpy(arrh_d, arrh, size, hipMemcpyHostToDevice);  
   sort<<<1, len-1>>>(arr_d, pivot, len, arrl_d, arrh_d);  
   hipMemcpy(arrl, arrl_d, size, hipMemcpyDeviceToHost);  
   hipMemcpy(arrh, arrh_d, size, hipMemcpyDeviceToHost);  
   int *temp1 = new int[len];  
   int *temp2 = new int[len];  
   for(int i=0; i<len; i++)  
   {  
     temp1[i]=temp2[i]=-9999;  
   }  
   int j=0, k=0;  
   for(int i=0; i<len; i++)  
   {  
     if(arrl[i]!=-9999)  
     {  
       temp1[j++] = arrl[i];  
     }  
     if(arrh[i]!=-9999)  
     {  
       temp2[k++] = arrh[i];  
     }  
   }  
   quicksort(temp1, j);  
   int p=0;  
   for(int i=0; i<j; i++)  
     arr[p++] = temp1[i];  
   arr[p++] = pivot;  
   quicksort(temp2, k);  
   for(int i=0; i<k; i++)  
     arr[p++] = temp2[i];  
   delete(arrl); delete(arrh); delete(temp1); delete(temp2);  
   hipFree(arr_d); hipFree(arrl_d); hipFree(arrh_d);  
 }  
 int main()  
 {  
   int n;  
   cout<<"\nEnter no. of elements you want to sort: ";  
   cin>>n;  
   int arr[n];  
   cout<<"\n\nEnter no.s to be sorted: \n";  
   for (int i = 0 ; i < n ; i++)  
     cin>>arr[i];  
   quicksort(arr, n);  
   cout<<"\nSorted array is: \n";  
   for(int i=0;i<n;i++)  
       cout<<arr[i]<<"\t";  
   return 0;  
 }  

