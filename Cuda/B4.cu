#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void mul(int *a, int n)
{
  __shared__ int s[4];
  int t = threadIdx.x;

  s[t] = a[2*t]*a[2*t+1];
  a[2*t]=s[t];
}

int main(void)
{
  const int n = 8;
  int a[n], d[n],ans;

  int no,x,y;
  cout <<"Enter your number" << endl;
  cin>>no;

  x=(no/10)*10;
  y=no%10;
  cout<<"x:"<<x<<" y:"<<y<<endl;
  a[0]=a[1]=a[2]=a[4]=x;
  a[3]=a[5]=a[6]=a[7]=y;

  int *d_d;
  hipMalloc(&d_d, n * sizeof(int));
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  mul<<<1,n/2>>>(d_d, n/2);
  hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_d);
  ans=d[0]+d[2]+d[4]+d[6];

  cout<<"The Square is:"<<ans;

 return 0;
}


