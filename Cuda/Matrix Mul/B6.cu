#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
using namespace std;

__global__ void matrixMulShared(int* A, int* B, int* C, int n)
{
	__shared__ int sA[8][8];//allocate shared memory per block
	__shared__ int sB[8][8];

	//find Row and Column corresponding to a data element for each thread
	int Row = blockDim.y*blockIdx.y + threadIdx.y;
	int Col = blockDim.x*blockIdx.x + threadIdx.x;
	int Cvalue = 0;

	sA[threadIdx.y][threadIdx.x] = 0;
	sB[threadIdx.y][threadIdx.x] = 0;

	//iterate through TILEs to traverse whole WIDTH tile = blockdim.x here 8
	for (int k = 0; k<(((n - 1) / 8) + 1); k++)
	{
		// copy values of data TILE into shared memory
		if (Row<n && (threadIdx.x + (k * 8))<n)
			sA[threadIdx.y][threadIdx.x] = A[(Row*n) + threadIdx.x + (k * 8)];
		else
			sA[threadIdx.y][threadIdx.x] = 0;
		if (Col<n && (threadIdx.y + (k * 8))<n)
			sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + (k * 8))*n + Col];
		else
			sB[threadIdx.y][threadIdx.x] = 0;
		//synchronize to confirm that whole partial product corresponding to all threads of the block has been calculated
		__syncthreads();

		for (int j = 0; j<8; ++j)
			Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
	}
	if (Row<n && Col<n)
		C[Row*n + Col] = Cvalue;
}
void matMulOnHost(int* A, int* B, int* C, int n)
{
	for (int i = 0; i<n; i++)
		for (int j = 0; j<n; j++)
		{
			C[i*n + j] = 0;
			for (int k = 0; k<n; k++)
				C[i*n + j] += A[i*n + k] * B[k*n + j];
		}
	return;
}
int main(int argc, char ** argv)
{
	
	int *hostA, *hostB, *hostC, *hC, *devA, *devB, *devC, n;
	cout << "Enter n: ";
	cin >> n;

	//allocate host side memory
	hostA = (int*)malloc(sizeof(int)*n*n);
	hostB = (int*)malloc(sizeof(int)*n*n);
	hostC = (int*)malloc(sizeof(int)*n*n);
	hC = (int*)malloc(sizeof(int)*n*n);


	for (int i = 0; i<n*n; i++)
		hostA[i] = rand() % 10;
	for (int i = 0; i<n*n; i++)
		hostB[i] = rand() % 10;
	

	//allocate device memory
	hipMalloc((void **)&devA, sizeof(int)*n*n);
	hipMalloc((void **)&devB, sizeof(int)*n*n);
	hipMalloc((void **)&devC, sizeof(int)*n*n);

	clock_t begin = clock();

	matMulOnHost(hostA, hostB, hC, n);

	clock_t end = clock();
	double time2 = (double)(end - begin) / CLOCKS_PER_SEC;


	//copy value from host to device
	hipMemcpy(devA, hostA, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(devB, hostB, sizeof(int)*n*n, hipMemcpyHostToDevice);
	//calculate execution configuration
	dim3 dimBlock(8, 8, 1);
	dim3 dimGrid((n / 8) + 1, (n / 8) + 1, 1);//creating just sufficient no of blocks


	
	float time1;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	matrixMulShared << <dimGrid, dimBlock >> >(devA, devB, devC, n);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	//time taken in kernel call calculated
	hipEventElapsedTime(&time1, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	

	hipDeviceSynchronize();
	hipMemcpy(hostC, devC, sizeof(int)*n*n, hipMemcpyDeviceToHost);

	cout << "\nMatrix A:\n";
	for (int i = 0, k = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++, k++)
			cout << hostA[k] << "\t";
		cout << endl;
	}
	cout << "\nMatrix B:\n";
	for (int i = 0, k = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++, k++)
			cout << hostB[k] << "\t";
		cout << endl;
	}
	cout << "\nMatrix multiplication using shared memory:\n";
	for (int i = 0, k = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++, k++)
			cout << hostC[k] << "\t";
		cout << endl;
	}
	cout << "\nMatrix multiplication on host:\n";
	for (int i = 0, k = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++, k++)
			cout << hC[k] << "\t";
		cout << endl;
	}
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	free(hostA);
	free(hostB);
	free(hostC);
	free(hC);


	
	printf("\n\nTime taken by cuda is %f (ms)\n", time1);
	time2= time2 * 1000;
	printf("\n\nTime taken by host is %f (ms)\n", time2);
	return 0;
}